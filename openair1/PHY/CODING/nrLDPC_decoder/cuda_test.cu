#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_kernel(int id) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello from GPU kernel, id=%d\n", id);
    }
}

int main() {
    // 查询 GPU 属性
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("Using device %d: %s (compute capability %d.%d)\n",
           device, prop.name, prop.major, prop.minor);

    // 启动 kernel
    hello_kernel<<<1,1>>>(42);

    // 检查 launch 是否成功
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // 同步，等待 GPU 执行完
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel execution error: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("Kernel finished successfully!\n");
    return 0;
}
