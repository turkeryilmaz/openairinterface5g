#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "oai_cuda.h"


#define CHECK_CUDA(val) checkCuda((val), #val, __FILE__, __LINE__)
static void checkCuda(hipError_t result, const char* const func, const char *const file, const int line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

__device__ __forceinline__ float2 complex_mul(float2 a, float2 b) {
    return make_float2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__device__ __forceinline__ float2 complex_add(float2 a, float2 b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

__global__ void interleave_output_kernel(const float* __restrict__ rx_re,
                                         const float* __restrict__ rx_im,
                                         float2* __restrict__ output_interleaved,
                                         int num_total_samples)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_total_samples) {
        output_interleaved[i].x = rx_re[i];
        output_interleaved[i].y = rx_im[i];
    }
}

__global__ void multipath_channel_kernel(
    const float2* __restrict__ d_channel_coeffs,
    // const float2* __restrict__ tx_sig,
    const float* __restrict__ tx_sig,
    float2* __restrict__ rx_sig,
    int num_samples,
    int channel_length,
    int nb_tx,
    int nb_rx,
    float path_loss)
{
    extern __shared__ float2 tx_shared[];
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int ii = blockIdx.y;

    if (i >= num_samples) return;

    float2 rx_tmp = make_float2(0.0f, 0.0f);

    for (int j = 0; j < nb_tx; j++) {
        const int tid = threadIdx.x;
        const int block_start_idx = blockIdx.x * blockDim.x;
        const int shared_mem_size = blockDim.x + channel_length - 1;

        for (int k = tid; k < shared_mem_size; k += blockDim.x) {
            int load_idx = block_start_idx + k - (channel_length - 1);
            if (load_idx >= 0 && load_idx < num_samples) {
                // tx_shared[k] = tx_sig[j * num_samples + load_idx];
                // --- CHANGED: Read two floats and construct a float2 ---
                int interleaved_idx = 2 * (j * num_samples + load_idx);
                tx_shared[k] = make_float2(tx_sig[interleaved_idx], tx_sig[interleaved_idx + 1]);
            } else {
                tx_shared[k] = make_float2(0.0f, 0.0f);
            }
        }
        __syncthreads();

        for (int l = 0; l < channel_length; l++) {
            float2 tx_sample = tx_shared[tid + (channel_length - 1) - l];
            int chan_link_idx = ii + (j * nb_rx);
            float2 chan_weight = d_channel_coeffs[chan_link_idx * channel_length + l];
            rx_tmp = complex_add(rx_tmp, complex_mul(tx_sample, chan_weight));
        }
        __syncthreads();
    }
    
    rx_sig[ii * num_samples + i].x = rx_tmp.x * path_loss;
    rx_sig[ii * num_samples + i].y = rx_tmp.y * path_loss;
}


__global__ void multipath_channel_kernel_batched(
    const float2* __restrict__ d_channel_coeffs,
    const float2* __restrict__ tx_sig,
    float2* __restrict__ rx_sig,
    int num_samples,
    int channel_length,
    int nb_tx,
    int nb_rx,
    const float* __restrict__ path_loss_batch)
{
    extern __shared__ float2 tx_shared[];
    

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int ii = blockIdx.y;                         
    const int c = blockIdx.z;                         

    if (i >= num_samples) return;

    float2 rx_tmp = make_float2(0.0f, 0.0f);
    const float path_loss = path_loss_batch[c]; 

    const int channel_tx_offset = c * nb_tx * num_samples;
    const int channel_rx_offset = c * nb_rx * num_samples;

    for (int j = 0; j < nb_tx; j++) {
        const int tid = threadIdx.x;
        const int block_start_idx = blockIdx.x * blockDim.x;
        const int shared_mem_size = blockDim.x + channel_length - 1;

        for (int k = tid; k < shared_mem_size; k += blockDim.x) {
            int load_idx = block_start_idx + k - (channel_length - 1);
            if (load_idx >= 0 && load_idx < num_samples) {
                tx_shared[k] = tx_sig[channel_tx_offset + j * num_samples + load_idx];
            } else {
                tx_shared[k] = make_float2(0.0f, 0.0f);
            }
        }
        __syncthreads();

        for (int l = 0; l < channel_length; l++) {
            float2 tx_sample = tx_shared[tid + (channel_length - 1) - l];
            int chan_link_idx = (c * nb_tx * nb_rx) + (ii + j * nb_rx);
            float2 chan_weight = d_channel_coeffs[chan_link_idx * channel_length + l];
            rx_tmp = complex_add(rx_tmp, complex_mul(tx_sample, chan_weight));
        }
        __syncthreads();
    }
    
    rx_sig[channel_rx_offset + ii * num_samples + i].x = rx_tmp.x * path_loss;
    rx_sig[channel_rx_offset + ii * num_samples + i].y = rx_tmp.y * path_loss;
}



extern "C" {

void multipath_channel_cuda(
    // float **tx_sig_re, float **tx_sig_im,
    float **tx_sig_interleaved,
    float **rx_sig_re, float **rx_sig_im,
    int nb_tx, int nb_rx, int channel_length,
    uint32_t length, uint64_t channel_offset,
    float path_loss,
    float *h_channel_coeffs,
    void *d_tx_sig_void, void *d_rx_sig_void,
    void *d_channel_coeffs_void,
    void *h_tx_sig_pinned_void 
)
{
    // float2 *d_tx_sig = (float2*)d_tx_sig_void;
    float *d_tx_sig = (float*)d_tx_sig_void;
    float2 *d_rx_sig = (float2*)d_rx_sig_void;
    float2 *d_channel_coeffs = (float2*)d_channel_coeffs_void;
    int num_samples = length - (int)channel_offset;
    // float2* kernel_input_ptr;
    float* kernel_input_ptr;

    #if defined(USE_UNIFIED_MEMORY)
            // for (int j = 0; j < nb_tx; j++) {
            //     for (int i = 0; i < num_samples; i++) {
            //         d_tx_sig[j * num_samples + i] = make_float2(tx_sig_re[j][i], tx_sig_im[j][i]);
            //     }
            // }
            // For UM, we can just copy the host data into the managed buffer
            for (int j = 0; j < nb_tx; j++) {
                memcpy(d_tx_sig + j * num_samples * 2, tx_sig_interleaved[j], num_samples * 2 * sizeof(float));
            }
            kernel_input_ptr = d_tx_sig;
    #elif defined(USE_ATS_MEMORY)
            // float2* h_tx_sig_pinned = (float2*)h_tx_sig_pinned_void;
            float* h_tx_sig_pinned = (float*)h_tx_sig_pinned_void;
            for (int j = 0; j < nb_tx; j++) {
                // for (int i = 0; i < num_samples; i++) {
                //     h_tx_sig_pinned[j * num_samples + i] = make_float2(tx_sig_re[j][i], tx_sig_im[j][i]);
                // }
                memcpy(h_tx_sig_pinned + j * num_samples * 2, tx_sig_interleaved[j], num_samples * 2 * sizeof(float));
            }
            kernel_input_ptr = h_tx_sig_pinned; 
    #else // EXPLICIT COPY
            // float2* h_tx_sig_pinned = (float2*)h_tx_sig_pinned_void;
            float* h_tx_sig_pinned = (float*)h_tx_sig_pinned_void;
            for (int j = 0; j < nb_tx; j++) {
                // for (int i = 0; i < num_samples; i++) {
                //     h_tx_sig_pinned[j * num_samples + i] = make_float2(tx_sig_re[j][i], tx_sig_im[j][i]);
                // }
            // }
            // CHECK_CUDA( hipMemcpy(d_tx_sig, h_tx_sig_pinned, nb_tx * num_samples * sizeof(float2), hipMemcpyHostToDevice) );
            memcpy(h_tx_sig_pinned + j * num_samples * 2, tx_sig_interleaved[j], num_samples * 2 * sizeof(float));
            }
            CHECK_CUDA( hipMemcpy(d_tx_sig, h_tx_sig_pinned, nb_tx * num_samples * 2 * sizeof(float), hipMemcpyHostToDevice) );
            
            kernel_input_ptr = d_tx_sig;
    #endif

    size_t channel_size_bytes = nb_tx * nb_rx * channel_length * sizeof(float2);
    CHECK_CUDA( hipMemcpy(d_channel_coeffs, h_channel_coeffs, channel_size_bytes, hipMemcpyHostToDevice) );

    dim3 threadsPerBlock(512, 1);
    dim3 numBlocks((num_samples + threadsPerBlock.x - 1) / threadsPerBlock.x, nb_rx);
    size_t sharedMemSize = (threadsPerBlock.x + channel_length - 1) * sizeof(float2);
    multipath_channel_kernel<<<numBlocks, threadsPerBlock, sharedMemSize>>>(
        d_channel_coeffs, kernel_input_ptr, d_rx_sig, num_samples, channel_length, nb_tx, nb_rx, path_loss);
         
    #if defined(USE_UNIFIED_MEMORY)
            CHECK_CUDA( hipDeviceSynchronize() );
            for (int ii = 0; ii < nb_rx; ii++) {
                for (int i = 0; i < num_samples; i++) {
                    float2 result = d_rx_sig[ii * num_samples + i];
                    rx_sig_re[ii][i + channel_offset] = result.x;
                    rx_sig_im[ii][i + channel_offset] = result.y;
                }
            }
    #else
            CHECK_CUDA( hipDeviceSynchronize() ); 
            float2* h_rx_sig = (float2*)malloc(nb_rx * num_samples * sizeof(float2));
            CHECK_CUDA( hipMemcpy(h_rx_sig, d_rx_sig, nb_rx * num_samples * sizeof(float2), hipMemcpyDeviceToHost) );
            for (int ii = 0; ii < nb_rx; ii++) {
                for (int i = 0; i < num_samples; i++) {
                    float2 result = h_rx_sig[ii * num_samples + i];
                    rx_sig_re[ii][i + channel_offset] = result.x;
                    rx_sig_im[ii][i + channel_offset] = result.y;
                }
            }
            free(h_rx_sig);
    #endif
}


void interleave_channel_output_cuda(float **rx_sig_re,
                                    float **rx_sig_im,
                                    float2 **output_interleaved,
                                    int nb_rx,
                                    int num_samples)
{
    int num_total_samples = nb_rx * num_samples;
    size_t total_bytes_per_plane = num_total_samples * sizeof(float);
    size_t total_bytes_interleaved = num_total_samples * sizeof(float2);

    float *h_re_flat = (float*)malloc(total_bytes_per_plane);
    float *h_im_flat = (float*)malloc(total_bytes_per_plane);
    float2 *h_out_flat = (float2*)malloc(total_bytes_interleaved);

    for (int i = 0; i < nb_rx; i++) {
        memcpy(h_re_flat + i * num_samples, rx_sig_re[i], num_samples * sizeof(float));
        memcpy(h_im_flat + i * num_samples, rx_sig_im[i], num_samples * sizeof(float));
    }
    float *d_re, *d_im;
    float2 *d_out;
    CHECK_CUDA( hipMalloc((void**)&d_re, total_bytes_per_plane) );
    CHECK_CUDA( hipMalloc((void**)&d_im, total_bytes_per_plane) );
    CHECK_CUDA( hipMalloc((void**)&d_out, total_bytes_interleaved) );
    CHECK_CUDA( hipMemcpy(d_re, h_re_flat, total_bytes_per_plane, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(d_im, h_im_flat, total_bytes_per_plane, hipMemcpyHostToDevice) );

    int threadsPerBlock = 512;
    int blocksPerGrid = (num_total_samples + threadsPerBlock - 1) / threadsPerBlock;
    interleave_output_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_re, d_im, d_out, num_total_samples);

    CHECK_CUDA( hipMemcpy(h_out_flat, d_out, total_bytes_interleaved, hipMemcpyDeviceToHost) );
    
    for (int i = 0; i < nb_rx; i++) {
        memcpy(output_interleaved[i], h_out_flat + i * num_samples, num_samples * sizeof(float2));
    }
    free(h_re_flat);
    free(h_im_flat);
    free(h_out_flat);
    CHECK_CUDA( hipFree(d_re) );
    CHECK_CUDA( hipFree(d_im) );
    CHECK_CUDA( hipFree(d_out) );
}

} // extern "C"